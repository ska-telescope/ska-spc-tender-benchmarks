#include "hip/hip_runtime.h"
#include "common_gridding.cuh"
#include "gridding_benchmark.h"
#include "gridding_configuration.h"
#include "math.cuh"
#include "math.hpp"
#include "util.cuh"

// #define ALIGN(N, A) (((N) + (A)-1) / (A) * (A))
#define NUM_THREADS 128
#define UNROLL_PIXELS 4
#define BATCH_SIZE 128
#define MAX_NR_CHANNELS 8

#define V7

namespace gridding_benchmark
{
    template <typename T>
    __device__ inline void apply_aterm_gridder_(T *pixels, const T *aterm1, const T *aterm2)
    {
        // Aterm 1 hermitian
        T aterm1_h[4];
        hermitian(aterm1, aterm1_h);

        // Apply aterm: P = A1^H * P
        T temp[4];
        matmul(aterm1_h, pixels, temp);

        // Apply aterm: P = P * A2
        matmul(temp, aterm2, pixels);
    }

#ifdef V1
    __global__ void kernel_gridder(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                                   int nchannels, int nstations, const UVWCoordinate<float> *uvw,
                                   const float *wavenumbers, const float2 *visibilities, const float *spheroidal,
                                   const float2 *aterms, const Metadata *metadata, float2 *subgrids)
    {
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        int tid = tidx + tidy * blockDim.x;
        int nr_threads = blockDim.x * blockDim.y;
        int s = blockIdx.x;

        // Find offset of first subgrid
        const Metadata m_0 = metadata[0];
        const int baseline_offset_1 = m_0.baseline_offset;

        // Load metadata
        const Metadata m = metadata[s];
        const int time_offset = (m.baseline_offset - baseline_offset_1) + m.time_offset;
        const int nr_timesteps = m.nr_timesteps;
        const int aterm_index = m.aterm_index;
        const int station1 = m.baseline.station1;
        const int station2 = m.baseline.station2;
        const int x_coordinate = m.coordinate.x;
        const int y_coordinate = m.coordinate.y;
        const float w_offset_in_lambda = w_step_in_lambda * (m.coordinate.z + 0.5);

        // Compute u and v offset in wavelenghts
        const float u_offset = (x_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float v_offset = (y_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float w_offset = 2 * M_PI * w_offset_in_lambda;

        // Iterate all pixels in subgrid
        for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads)
        {
            // for (int x = 0; x < subgrid_size; x++) {
            //  Initialize pixel for every polarization
            int x = i % subgrid_size;
            int y = i / subgrid_size;

            float2 pixels[n_correlations];
            for (int k = 0; k < n_correlations; k++)
            {
                pixels[k] = make_float2(0, 0);
            }

            // Compute l,m,n
            float l = compute_l(x, subgrid_size, image_size);
            float m = compute_m(y, subgrid_size, image_size);
            float n = compute_n(l, m);
            // Iterate all timesteps
            for (int time = 0; time < nr_timesteps; time++)
            {
                // Load UVW coordinates
                float u = uvw[time_offset + time].u;
                float v = uvw[time_offset + time].v;
                float w = uvw[time_offset + time].w;

                // Compute phase index
                float phase_index = u * l + v * m + w * n;

                // Compute phase offset
                float phase_offset = u_offset * l + v_offset * m + w_offset * n;

                // Update pixel for every channel
                for (int chan = 0; chan < nchannels; chan++)
                {
                    // Compute phase
                    float phase = phase_offset - (phase_index * wavenumbers[chan]);

                    // Compute phasor
                    float2 phasor = make_float2(cosf(phase), sinf(phase));

                    // Update pixel for every polarization

                    size_t index = (time_offset + time) * nchannels + chan;
                    for (int pol = 0; pol < n_correlations; pol++)
                    {
                        float2 visibility = visibilities[index * n_correlations + pol];
                        pixels[pol] += visibility * phasor;
                    }
                }
            }

            // Load a term for station1
            int station1_index = (aterm_index * nstations + station1) * subgrid_size * subgrid_size * n_correlations +
                                 y * subgrid_size * n_correlations + x * n_correlations;
            const float2 *aterm1_ptr = &aterms[station1_index];

            // Load aterm for station2
            int station2_index = (aterm_index * nstations + station2) * subgrid_size * subgrid_size * n_correlations +
                                 y * subgrid_size * n_correlations + x * n_correlations;
            const float2 *aterm2_ptr = &aterms[station2_index];
            // Apply aterm
            apply_aterm_gridder_(pixels, aterm1_ptr, aterm2_ptr);

            // Load spheroidal
            float sph = spheroidal[y * subgrid_size + x];

            // Set subgrid value
            for (int pol = 0; pol < n_correlations; pol++)
            {
                unsigned idx_subgrid = s * n_correlations * subgrid_size * subgrid_size +
                                       pol * subgrid_size * subgrid_size + y * subgrid_size + x;
                subgrids[idx_subgrid] = pixels[pol] * sph;
            }
            // }
        }
    }
#endif

#ifdef V2
    __global__ void kernel_gridder(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                                   int nr_channels, // channel_offset? for the macro?
                                   int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers, float2 *visibilities,
                                   float *spheroidal, float2 *aterms, Metadata *metadata, float2 *subgrids)
    {
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        int tid = tidx + tidy * blockDim.x;
        int nr_threads = blockDim.x * blockDim.y;
        int s = blockIdx.x;

        // Find offset of first subgrid
        const Metadata m_0 = metadata[0];
        const int baseline_offset_1 = m_0.baseline_offset;

        // Load metadata
        const Metadata m = metadata[s];
        const int time_offset = (m.baseline_offset - baseline_offset_1) + m.time_offset;
        const int nr_timesteps = m.nr_timesteps;
        const int aterm_index = m.aterm_index;
        const int station1 = m.baseline.station1;
        const int station2 = m.baseline.station2;
        const int x_coordinate = m.coordinate.x;
        const int y_coordinate = m.coordinate.y;
        const float w_offset_in_lambda = w_step_in_lambda * (m.coordinate.z + 0.5);

        // Compute u and v offset in wavelenghts
        const float u_offset = (x_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float v_offset = (y_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float w_offset = 2 * M_PI * w_offset_in_lambda;

        // Iterate all pixels in subgrid
        for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads)
        {
            // for (int x = 0; x < subgrid_size; x++) {
            //  Initialize pixel for every polarization
            int x = i % subgrid_size;
            int y = i / subgrid_size;

            float2 pixels[n_correlations];
            for (int k = 0; k < n_correlations; k++)
            {
                pixels[k] = make_float2(0, 0);
            }

            // Compute l,m,n
            float l = compute_l(x, subgrid_size, image_size);
            float m = compute_m(y, subgrid_size, image_size);
            float n = compute_n(l, m);
            // Iterate all timesteps
            for (int time = 0; time < nr_timesteps; time++)
            {
                // Load UVW coordinates
                float u = uvw[time_offset + time].u;
                float v = uvw[time_offset + time].v;
                float w = uvw[time_offset + time].w;

                // Compute phase index
                float phase_index = u * l + v * m + w * n;

                // Compute phase offset
                float phase_offset = u_offset * l + v_offset * m + w_offset * n;

                // Update pixel for every channel
                for (int chan = 0; chan < nr_channels; chan++)
                {
                    // Compute phase
                    float phase = phase_offset - (phase_index * wavenumbers[chan]);

                    // Compute phasor
                    float2 phasor = make_float2(__cosf(phase), __sinf(phase));

                    // Update pixel for every polarization

                    size_t index = (time_offset + time) * nr_channels + chan;
                    for (int pol = 0; pol < n_correlations; pol++)
                    {
                        float2 visibility = visibilities[index * n_correlations + pol];
                        pixels[pol] += visibility * phasor;
                    }
                }
            }

            // Load a term for station1
            int station1_index = (aterm_index * nr_stations + station1) * subgrid_size * subgrid_size * n_correlations +
                                 y * subgrid_size * n_correlations + x * n_correlations;
            float2 *aterm1_ptr = &aterms[station1_index];

            // Load aterm for station2
            int station2_index = (aterm_index * nr_stations + station2) * subgrid_size * subgrid_size * n_correlations +
                                 y * subgrid_size * n_correlations + x * n_correlations;
            float2 *aterm2_ptr = &aterms[station2_index];
            // Apply aterm
            apply_aterm_gridder_(pixels, aterm1_ptr, aterm2_ptr);

            // Load spheroidal
            float sph = spheroidal[y * subgrid_size + x];

            // Set subgrid value
            for (int pol = 0; pol < n_correlations; pol++)
            {
                unsigned idx_subgrid = s * n_correlations * subgrid_size * subgrid_size +
                                       pol * subgrid_size * subgrid_size + y * subgrid_size + x;
                subgrids[idx_subgrid] = pixels[pol] * sph;
            }
            // }
        }
    }
#endif

#ifdef V3
    __global__ void kernel_gridder(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                                   int nr_channels, int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers,
                                   float2 *visibilities, float *spheroidal, float2 *aterms, Metadata *metadata,
                                   float2 *subgrids)
    {

        int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        int tid = tidx + tidy * blockDim.x;
        int nr_threads = blockDim.x * blockDim.y;
        int s = blockIdx.x;

        // Find offset of first subgrid
        const Metadata m_0 = metadata[0];
        const int baseline_offset_1 = m_0.baseline_offset;

        // Load metadata
        const Metadata m = metadata[s];
        const int time_offset_global = (m.baseline_offset - baseline_offset_1) + m.time_offset;
        const int nr_timesteps = m.nr_timesteps;
        const int aterm_index = m.aterm_index;
        const int station1 = m.baseline.station1;
        const int station2 = m.baseline.station2;
        const int x_coordinate = m.coordinate.x;
        const int y_coordinate = m.coordinate.y;
        const float w_offset_in_lambda = w_step_in_lambda * (m.coordinate.z + 0.5);

        // Compute u and v offset in wavelenghts
        const float u_offset = (x_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float v_offset = (y_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float w_offset = 2 * M_PI * w_offset_in_lambda;

        // Iterate all pixels in subgrid
        for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads)
        {
            // for (int x = 0; x < subgrid_size; x++) {
            //  Initialize pixel for every polarization
            int x = i % subgrid_size;
            int y = i / subgrid_size;

            float2 pixelXX;
            float2 pixelXY;
            float2 pixelYX;
            float2 pixelYY;

            pixelXX = make_float2(0, 0);
            pixelXY = make_float2(0, 0);
            pixelYX = make_float2(0, 0);
            pixelYY = make_float2(0, 0);

            float l = compute_l(x, subgrid_size, image_size);
            float m = compute_m(y, subgrid_size, image_size);
            float n = compute_n(l, m);

            // Iterate all timesteps
            for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local++)
            {
                // Load visibilities

                float u = uvw[time_offset_global + time_offset_local].u;
                float v = uvw[time_offset_global + time_offset_local].v;
                float w = uvw[time_offset_global + time_offset_local].w;

                // Compute phase index
                float phase_index = u * l + v * m + w * n;

                float phase_offset = u_offset * l + v_offset * m + w_offset * n;

                for (int chan = 0; chan < nr_channels; chan++)
                {
                    // Compute phase
                    float phase = phase_offset - (phase_index * wavenumbers[chan]);

                    // Compute phasor
                    float2 phasor = make_float2(__cosf(phase), __sinf(phase));

                    // Update pixel for every polarization

                    int idx_time = time_offset_global + time_offset_local;

                    int indexXX = index_visibility(nr_channels, idx_time, chan, 0);
                    int indexXY = index_visibility(nr_channels, idx_time, chan, 1);
                    int indexYX = index_visibility(nr_channels, idx_time, chan, 2);
                    int indexYY = index_visibility(nr_channels, idx_time, chan, 3);

                    float2 visXX = visibilities[indexXX];
                    float2 visXY = visibilities[indexXY];
                    float2 visYX = visibilities[indexYX];
                    float2 visYY = visibilities[indexYY];

                    pixelXX.x += phasor.x * visXX.x;
                    pixelXX.y += phasor.x * visXX.y;
                    pixelXX.x -= phasor.y * visXX.y;
                    pixelXX.y += phasor.y * visXX.x;

                    pixelXY.x += phasor.x * visXY.x;
                    pixelXY.y += phasor.x * visXY.y;
                    pixelXY.x -= phasor.y * visXY.y;
                    pixelXY.y += phasor.y * visXY.x;

                    pixelYX.x += phasor.x * visYX.x;
                    pixelYX.y += phasor.x * visYX.y;
                    pixelYX.x -= phasor.y * visYX.y;
                    pixelYX.y += phasor.y * visYX.x;

                    pixelYY.x += phasor.x * visYY.x;
                    pixelYY.y += phasor.x * visYY.y;
                    pixelYY.x -= phasor.y * visYY.y;
                    pixelYY.y += phasor.y * visYY.x;
                }
            }

            // Load aterm for station1
            float2 aXX1, aXY1, aYX1, aYY1;
            read_aterm(subgrid_size, nr_stations, aterm_index, station1, y, x, aterms, &aXX1, &aXY1, &aYX1, &aYY1);

            // Load aterm for station2
            float2 aXX2, aXY2, aYX2, aYY2;
            read_aterm(subgrid_size, nr_stations, aterm_index, station2, y, x, aterms, &aXX2, &aXY2, &aYX2, &aYY2);

            // Apply the conjugate transpose of the A-term
            apply_aterm(conj(aXX1), conj(aYX1), conj(aXY1), conj(aYY1), conj(aXX2), conj(aYX2), conj(aXY2), conj(aYY2),
                        pixelXX, pixelXY, pixelYX, pixelYY);

            // Load a term for station1
            // Load spheroidal
            float sph = spheroidal[y * subgrid_size + x];

            int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i);
            int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i);
            int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i);
            int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i);

            subgrids[idx_xx] += pixelXX * sph;
            subgrids[idx_xy] += pixelXY * sph;
            subgrids[idx_yx] += pixelYX * sph;
            subgrids[idx_yy] += pixelYY * sph;

            // }
        }
    }
#endif

#ifdef V4
    __shared__ float wavenumbers_v4_[MAX_NR_CHANNELS];

    template <int current_nr_channels>
    __device__ void kernel_gridder_(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                                    int nr_channels,
                                    int channel_offset, // channel_offset? for the macro?
                                    int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers,
                                    float2 *visibilities, float *spheroidal, float2 *aterms, Metadata *metadata,
                                    float2 *subgrids)
    {
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        int tid = tidx + tidy * blockDim.x;
        int nr_threads = blockDim.x * blockDim.y;
        int s = blockIdx.x;

        // Find offset of first subgrid
        const Metadata m_0 = metadata[0];
        const int baseline_offset_1 = m_0.baseline_offset;

        // Load metadata
        const Metadata m = metadata[s];
        const int time_offset_global = (m.baseline_offset - baseline_offset_1) + m.time_offset;
        const int nr_timesteps = m.nr_timesteps;
        const int aterm_index = m.aterm_index;
        const int station1 = m.baseline.station1;
        const int station2 = m.baseline.station2;
        const int x_coordinate = m.coordinate.x;
        const int y_coordinate = m.coordinate.y;
        const float w_offset_in_lambda = w_step_in_lambda * (m.coordinate.z + 0.5);

        // Set subgrid to zero
        if (channel_offset == 0)
        {
            for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads)
            {
                int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i);
                int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i);
                int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i);
                int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i);
                subgrids[idx_xx] = make_float2(0, 0);
                subgrids[idx_xy] = make_float2(0, 0);
                subgrids[idx_yx] = make_float2(0, 0);
                subgrids[idx_yy] = make_float2(0, 0);
            }
        }

        for (int i = tid; i < current_nr_channels; i += nr_threads)
        {
            wavenumbers_v4_[i] = wavenumbers[i + channel_offset];
        }

        __syncthreads();

        // Compute u and v offset in wavelenghts
        const float u_offset = (x_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float v_offset = (y_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float w_offset = 2 * M_PI * w_offset_in_lambda;

        // Iterate all pixels in subgrid
        for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads)
        {
            // for (int x = 0; x < subgrid_size; x++) {
            //  Initialize pixel for every polarization
            int x = i % subgrid_size;
            int y = i / subgrid_size;

            float2 pixelXX;
            float2 pixelXY;
            float2 pixelYX;
            float2 pixelYY;

            pixelXX = make_float2(0, 0);
            pixelXY = make_float2(0, 0);
            pixelYX = make_float2(0, 0);
            pixelYY = make_float2(0, 0);

            float l = compute_l(x, subgrid_size, image_size);
            float m = compute_m(y, subgrid_size, image_size);
            float n = compute_n(l, m);

            // Iterate all timesteps
            for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local++)
            {
                // Load visibilities

                float u = uvw[time_offset_global + time_offset_local].u;
                float v = uvw[time_offset_global + time_offset_local].v;
                float w = uvw[time_offset_global + time_offset_local].w;

                // Compute phase index
                float phase_index = u * l + v * m + w * n;

                float phase_offset = u_offset * l + v_offset * m + w_offset * n;

                for (int chan = 0; chan < current_nr_channels; chan++)
                {
                    // Compute phase
                    float phase = phase_offset - (phase_index * wavenumbers_v4_[chan]);

                    // Compute phasor
                    float2 phasor = make_float2(__cosf(phase), __sinf(phase));

                    // Update pixel for every polarization

                    int idx_time = time_offset_global + time_offset_local;
                    int idx_chan = channel_offset + chan;

                    int indexXX = index_visibility(nr_channels, idx_time, idx_chan, 0);
                    int indexXY = index_visibility(nr_channels, idx_time, idx_chan, 1);
                    int indexYX = index_visibility(nr_channels, idx_time, idx_chan, 2);
                    int indexYY = index_visibility(nr_channels, idx_time, idx_chan, 3);

                    float2 visXX = visibilities[indexXX];
                    float2 visXY = visibilities[indexXY];
                    float2 visYX = visibilities[indexYX];
                    float2 visYY = visibilities[indexYY];

                    pixelXX.x += phasor.x * visXX.x;
                    pixelXX.y += phasor.x * visXX.y;
                    pixelXX.x -= phasor.y * visXX.y;
                    pixelXX.y += phasor.y * visXX.x;

                    pixelXY.x += phasor.x * visXY.x;
                    pixelXY.y += phasor.x * visXY.y;
                    pixelXY.x -= phasor.y * visXY.y;
                    pixelXY.y += phasor.y * visXY.x;

                    pixelYX.x += phasor.x * visYX.x;
                    pixelYX.y += phasor.x * visYX.y;
                    pixelYX.x -= phasor.y * visYX.y;
                    pixelYX.y += phasor.y * visYX.x;

                    pixelYY.x += phasor.x * visYY.x;
                    pixelYY.y += phasor.x * visYY.y;
                    pixelYY.x -= phasor.y * visYY.y;
                    pixelYY.y += phasor.y * visYY.x;
                }
            }

            // Load aterm for station1
            float2 aXX1, aXY1, aYX1, aYY1;
            read_aterm(subgrid_size, nr_stations, aterm_index, station1, y, x, aterms, &aXX1, &aXY1, &aYX1, &aYY1);

            // Load aterm for station2
            float2 aXX2, aXY2, aYX2, aYY2;
            read_aterm(subgrid_size, nr_stations, aterm_index, station2, y, x, aterms, &aXX2, &aXY2, &aYX2, &aYY2);

            // Apply the conjugate transpose of the A-term
            apply_aterm(conj(aXX1), conj(aYX1), conj(aXY1), conj(aYY1), conj(aXX2), conj(aYX2), conj(aXY2), conj(aYY2),
                        pixelXX, pixelXY, pixelYX, pixelYY);

            // Load a term for station1
            // Load spheroidal
            float sph = spheroidal[y * subgrid_size + x];

            int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i);
            int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i);
            int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i);
            int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i);

            subgrids[idx_xx] += pixelXX * sph;
            subgrids[idx_xy] += pixelXY * sph;
            subgrids[idx_yx] += pixelYX * sph;
            subgrids[idx_yy] += pixelYY * sph;

            // }
        }
    }

#define KERNEL_GRIDDER_TEMPLATE(current_nr_channels)                                                                   \
    for (; (channel_offset + current_nr_channels) <= nr_channels; channel_offset += current_nr_channels)               \
    {                                                                                                                  \
        kernel_gridder_<current_nr_channels>(grid_size, subgrid_size, image_size, w_step_in_lambda, nr_channels,       \
                                             channel_offset, nr_stations, uvw, wavenumbers, visibilities, spheroidal,  \
                                             aterms, metadata, subgrids);                                              \
    }

    __global__ void

    kernel_gridder(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                   int nr_channels, // channel_offset? for the macro?
                   int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers, float2 *visibilities,
                   float *spheroidal, float2 *aterms, Metadata *metadata, float2 *subgrids)
    {
        int channel_offset = 0;
        KERNEL_GRIDDER_TEMPLATE(8);
        KERNEL_GRIDDER_TEMPLATE(7);
        KERNEL_GRIDDER_TEMPLATE(6);
        KERNEL_GRIDDER_TEMPLATE(5);
        KERNEL_GRIDDER_TEMPLATE(4);
        KERNEL_GRIDDER_TEMPLATE(3);
        KERNEL_GRIDDER_TEMPLATE(2);
        KERNEL_GRIDDER_TEMPLATE(1);
    }
#endif

#ifdef V5
    __shared__ float2 visibilities_v5_[BATCH_SIZE][n_correlations];
    __shared__ float4 uvw_v5_[BATCH_SIZE];
    __shared__ float wavenumbers_v5_[MAX_NR_CHANNELS];

    template <int current_nr_channels>
    __device__ void kernel_gridder_v5_(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                                       int nr_channels,
                                       int channel_offset, // channel_offset? for the macro?
                                       int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers,
                                       float2 *visibilities, float *spheroidal, float2 *aterms, Metadata *metadata,
                                       float2 *subgrids)
    {
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        int tid = tidx + tidy * blockDim.x;
        int nr_threads = blockDim.x * blockDim.y;
        int s = blockIdx.x;

        // Find offset of first subgrid
        const Metadata m_0 = metadata[0];
        const int baseline_offset_1 = m_0.baseline_offset;

        // Load metadata
        const Metadata m = metadata[s];
        const int time_offset_global = (m.baseline_offset - baseline_offset_1) + m.time_offset;
        const int nr_timesteps = m.nr_timesteps;
        const int aterm_index = m.aterm_index;
        const int station1 = m.baseline.station1;
        const int station2 = m.baseline.station2;
        const int x_coordinate = m.coordinate.x;
        const int y_coordinate = m.coordinate.y;
        const float w_offset_in_lambda = w_step_in_lambda * (m.coordinate.z + 0.5);

        // Set subgrid to zero
        if (channel_offset == 0)
        {
            for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads)
            {
                int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i);
                int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i);
                int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i);
                int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i);
                subgrids[idx_xx] = make_float2(0, 0);
                subgrids[idx_xy] = make_float2(0, 0);
                subgrids[idx_yx] = make_float2(0, 0);
                subgrids[idx_yy] = make_float2(0, 0);
            }
        }

        for (int i = tid; i < current_nr_channels; i += nr_threads)
        {
            wavenumbers_v5_[i] = wavenumbers[i + channel_offset];
        }

        __syncthreads();

        // Compute u and v offset in wavelenghts
        const float u_offset = (x_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float v_offset = (y_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float w_offset = 2 * M_PI * w_offset_in_lambda;

        // Iterate all pixels in subgrid
        for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads)
        {
            // for (int x = 0; x < subgrid_size; x++) {
            //  Initialize pixel for every polarization
            int x = i % subgrid_size;
            int y = i / subgrid_size;

            float2 pixelXX;
            float2 pixelXY;
            float2 pixelYX;
            float2 pixelYY;

            pixelXX = make_float2(0, 0);
            pixelXY = make_float2(0, 0);
            pixelYX = make_float2(0, 0);
            pixelYY = make_float2(0, 0);

            float l = compute_l(x, subgrid_size, image_size);
            float m = compute_m(y, subgrid_size, image_size);
            float n = compute_n(l, m);

            int current_nr_timesteps = BATCH_SIZE / MAX_NR_CHANNELS;
            // Iterate all timesteps
            for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps)
            {
                current_nr_timesteps = nr_timesteps - time_offset_local < current_nr_timesteps
                                           ? nr_timesteps - time_offset_local
                                           : current_nr_timesteps;
                __syncthreads();
                for (int time = tid; time < current_nr_timesteps; time += nr_threads)
                {
                    UVWCoordinate<float> a = uvw[time_offset_global + time_offset_local + time];
                    uvw_v5_[time] = make_float4(a.u, a.v, a.w, 0);
                }

                // Load visibilities
                for (int k = tid; k < current_nr_timesteps * current_nr_channels; k += nr_threads)
                {
                    int idx_time = time_offset_global + time_offset_local + (k / current_nr_channels);
                    int idx_chan = channel_offset + (k % current_nr_channels);
                    int indexXX = index_visibility(nr_channels, idx_time, idx_chan, 0);
                    int indexXY = index_visibility(nr_channels, idx_time, idx_chan, 1);
                    int indexYX = index_visibility(nr_channels, idx_time, idx_chan, 2);
                    int indexYY = index_visibility(nr_channels, idx_time, idx_chan, 3);
                    visibilities_v5_[k][0] = visibilities[indexXX];
                    visibilities_v5_[k][1] = visibilities[indexXY];
                    visibilities_v5_[k][2] = visibilities[indexYX];
                    visibilities_v5_[k][3] = visibilities[indexYY];
                }
                __syncthreads();

                for (int time = 0; time < current_nr_timesteps; time++)
                {

                    // Load UVW coordinates
                    float u = uvw_v5_[time].x;
                    float v = uvw_v5_[time].y;
                    float w = uvw_v5_[time].z;

                    // Compute phase index
                    float phase_index = u * l + v * m + w * n;

                    float phase_offset = u_offset * l + v_offset * m + w_offset * n;

                    for (int chan = 0; chan < current_nr_channels; chan++)
                    {
                        // Compute phase
                        float phase = phase_offset - (phase_index * wavenumbers_v5_[chan]);

                        // Compute phasor
                        float2 phasor = make_float2(__cosf(phase), __sinf(phase));

                        // Update pixel for every polarization

                        float2 visXX = visibilities_v5_[time * current_nr_channels + chan][0];
                        float2 visXY = visibilities_v5_[time * current_nr_channels + chan][1];
                        float2 visYX = visibilities_v5_[time * current_nr_channels + chan][2];
                        float2 visYY = visibilities_v5_[time * current_nr_channels + chan][3];

                        pixelXX.x += phasor.x * visXX.x;
                        pixelXX.y += phasor.x * visXX.y;
                        pixelXX.x -= phasor.y * visXX.y;
                        pixelXX.y += phasor.y * visXX.x;

                        pixelXY.x += phasor.x * visXY.x;
                        pixelXY.y += phasor.x * visXY.y;
                        pixelXY.x -= phasor.y * visXY.y;
                        pixelXY.y += phasor.y * visXY.x;

                        pixelYX.x += phasor.x * visYX.x;
                        pixelYX.y += phasor.x * visYX.y;
                        pixelYX.x -= phasor.y * visYX.y;
                        pixelYX.y += phasor.y * visYX.x;

                        pixelYY.x += phasor.x * visYY.x;
                        pixelYY.y += phasor.x * visYY.y;
                        pixelYY.x -= phasor.y * visYY.y;
                        pixelYY.y += phasor.y * visYY.x;
                    }
                }
            }

            // Load aterm for station1
            float2 aXX1, aXY1, aYX1, aYY1;
            read_aterm(subgrid_size, nr_stations, aterm_index, station1, y, x, aterms, &aXX1, &aXY1, &aYX1, &aYY1);

            // Load aterm for station2
            float2 aXX2, aXY2, aYX2, aYY2;
            read_aterm(subgrid_size, nr_stations, aterm_index, station2, y, x, aterms, &aXX2, &aXY2, &aYX2, &aYY2);

            // Apply the conjugate transpose of the A-term
            apply_aterm(conj(aXX1), conj(aYX1), conj(aXY1), conj(aYY1), conj(aXX2), conj(aYX2), conj(aXY2), conj(aYY2),
                        pixelXX, pixelXY, pixelYX, pixelYY);

            // Load a term for station1
            // Load spheroidal
            float sph = spheroidal[y * subgrid_size + x];

            int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i);
            int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i);
            int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i);
            int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i);

            subgrids[idx_xx] += pixelXX * sph;
            subgrids[idx_xy] += pixelXY * sph;
            subgrids[idx_yx] += pixelYX * sph;
            subgrids[idx_yy] += pixelYY * sph;

            // }
        }
    }

#define KERNEL_GRIDDER_TEMPLATE(current_nr_channels)                                                                   \
    for (; (channel_offset + current_nr_channels) <= nr_channels; channel_offset += current_nr_channels)               \
    {                                                                                                                  \
        kernel_gridder_v5_<current_nr_channels>(grid_size, subgrid_size, image_size, w_step_in_lambda, nr_channels,    \
                                                channel_offset, nr_stations, uvw, wavenumbers, visibilities,           \
                                                spheroidal, aterms, metadata, subgrids);                               \
    }

    __global__ void

    kernel_gridder(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                   int nr_channels, // channel_offset? for the macro?
                   int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers, float2 *visibilities,
                   float *spheroidal, float2 *aterms, Metadata *metadata, float2 *subgrids)
    {
        int channel_offset = 0;
        KERNEL_GRIDDER_TEMPLATE(8);
        KERNEL_GRIDDER_TEMPLATE(7);
        KERNEL_GRIDDER_TEMPLATE(6);
        KERNEL_GRIDDER_TEMPLATE(5);
        KERNEL_GRIDDER_TEMPLATE(4);
        KERNEL_GRIDDER_TEMPLATE(3);
        KERNEL_GRIDDER_TEMPLATE(2);
        KERNEL_GRIDDER_TEMPLATE(1);
    }

#endif

#ifdef V6

    __shared__ float2 visibilities_v6_[BATCH_SIZE][n_correlations];
    __shared__ float4 uvw_v6_[BATCH_SIZE];
    __shared__ float wavenumbers_v6_[MAX_NR_CHANNELS];

    template <int current_nr_channels>
    __device__ void kernel_gridder_v6_(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                                       int nr_channels,
                                       int channel_offset, // channel_offset? for the macro?
                                       int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers,
                                       float2 *visibilities, float *spheroidal, float2 *aterms, Metadata *metadata,
                                       float2 *subgrids)
    {
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        int tid = tidx + tidy * blockDim.x;
        int nr_threads = blockDim.x * blockDim.y;
        int s = blockIdx.x;

        // Find offset of first subgrid
        const Metadata m_0 = metadata[0];
        const int baseline_offset_1 = m_0.baseline_offset;

        // Load metadata
        const Metadata m = metadata[s];
        const int time_offset_global = (m.baseline_offset - baseline_offset_1) + m.time_offset;
        const int nr_timesteps = m.nr_timesteps;
        const int aterm_index = m.aterm_index;
        const int station1 = m.baseline.station1;
        const int station2 = m.baseline.station2;
        const int x_coordinate = m.coordinate.x;
        const int y_coordinate = m.coordinate.y;
        const float w_offset_in_lambda = w_step_in_lambda * (m.coordinate.z + 0.5);

        // Set subgrid to zero
        if (channel_offset == 0)
        {
            for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads)
            {
                int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i);
                int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i);
                int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i);
                int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i);
                subgrids[idx_xx] = make_float2(0, 0);
                subgrids[idx_xy] = make_float2(0, 0);
                subgrids[idx_yx] = make_float2(0, 0);
                subgrids[idx_yy] = make_float2(0, 0);
            }
        }

        for (int i = tid; i < current_nr_channels; i += nr_threads)
        {
            wavenumbers_v6_[i] = wavenumbers[i + channel_offset];
        }

        __syncthreads();

        // Compute u and v offset in wavelenghts
        const float u_offset = (x_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float v_offset = (y_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float w_offset = 2 * M_PI * w_offset_in_lambda;

        // Iterate all pixels in subgrid
        for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads * UNROLL_PIXELS)
        {
            // for (int x = 0; x < subgrid_size; x++) {
            //  Initialize pixel for every polarization
            // int x = i % subgrid_size;
            // int y = i / subgrid_size;

            float2 pixelXX[UNROLL_PIXELS];
            float2 pixelXY[UNROLL_PIXELS];
            float2 pixelYX[UNROLL_PIXELS];
            float2 pixelYY[UNROLL_PIXELS];

            for (int p = 0; p < UNROLL_PIXELS; p++)
            {
                pixelXX[p] = make_float2(0, 0);
                pixelXY[p] = make_float2(0, 0);
                pixelYX[p] = make_float2(0, 0);
                pixelYY[p] = make_float2(0, 0);
            }

            float l[UNROLL_PIXELS];
            float m[UNROLL_PIXELS];
            float n[UNROLL_PIXELS];
            float phase_offset[UNROLL_PIXELS];
            for (int p = 0; p < UNROLL_PIXELS; p++)
            {

                int x = (i + p * nr_threads) % subgrid_size;
                int y = (i + p * nr_threads) / subgrid_size;
                l[p] = compute_l(x, subgrid_size, image_size);
                m[p] = compute_m(y, subgrid_size, image_size);
                n[p] = compute_n(l[p], m[p]);
                phase_offset[p] = u_offset * l[p] + v_offset * m[p] + w_offset * n[p];
            }

            int current_nr_timesteps = BATCH_SIZE / MAX_NR_CHANNELS;
            // Iterate all timesteps
            for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps)
            {
                current_nr_timesteps = nr_timesteps - time_offset_local < current_nr_timesteps
                                           ? nr_timesteps - time_offset_local
                                           : current_nr_timesteps;
                __syncthreads();
                for (int time = tid; time < current_nr_timesteps; time += nr_threads)
                {
                    UVWCoordinate<float> a = uvw[time_offset_global + time_offset_local + time];
                    uvw_v6_[time] = make_float4(a.u, a.v, a.w, 0);
                }

                // Load visibilities
                for (int k = tid; k < current_nr_timesteps * current_nr_channels; k += nr_threads)
                {
                    int idx_time = time_offset_global + time_offset_local + (k / current_nr_channels);
                    int idx_chan = channel_offset + (k % current_nr_channels);
                    int indexXX = index_visibility(nr_channels, idx_time, idx_chan, 0);
                    int indexXY = index_visibility(nr_channels, idx_time, idx_chan, 1);
                    int indexYX = index_visibility(nr_channels, idx_time, idx_chan, 2);
                    int indexYY = index_visibility(nr_channels, idx_time, idx_chan, 3);
                    visibilities_v6_[k][0] = visibilities[indexXX];
                    visibilities_v6_[k][1] = visibilities[indexXY];
                    visibilities_v6_[k][2] = visibilities[indexYX];
                    visibilities_v6_[k][3] = visibilities[indexYY];
                }
                __syncthreads();

                for (int time = 0; time < current_nr_timesteps; time++)
                {

                    // Load UVW coordinates
                    float u = uvw_v6_[time].x;
                    float v = uvw_v6_[time].y;
                    float w = uvw_v6_[time].z;
                    for (int p = 0; p < UNROLL_PIXELS; p++)
                    {
                        // Compute phase index
                        float phase_index = u * l[p] + v * m[p] + w * n[p];

                        for (int chan = 0; chan < current_nr_channels; chan++)
                        {
                            // Compute phase
                            float phase = phase_offset[p] - (phase_index * wavenumbers_v6_[chan]);

                            // Compute phasor
                            float2 phasor = make_float2(__cosf(phase), __sinf(phase));

                            // Update pixel for every polarization

                            float2 visXX = visibilities_v6_[time * current_nr_channels + chan][0];
                            float2 visXY = visibilities_v6_[time * current_nr_channels + chan][1];
                            float2 visYX = visibilities_v6_[time * current_nr_channels + chan][2];
                            float2 visYY = visibilities_v6_[time * current_nr_channels + chan][3];

                            pixelXX[p].x += phasor.x * visXX.x;
                            pixelXX[p].y += phasor.x * visXX.y;
                            pixelXX[p].x -= phasor.y * visXX.y;
                            pixelXX[p].y += phasor.y * visXX.x;

                            pixelXY[p].x += phasor.x * visXY.x;
                            pixelXY[p].y += phasor.x * visXY.y;
                            pixelXY[p].x -= phasor.y * visXY.y;
                            pixelXY[p].y += phasor.y * visXY.x;

                            pixelYX[p].x += phasor.x * visYX.x;
                            pixelYX[p].y += phasor.x * visYX.y;
                            pixelYX[p].x -= phasor.y * visYX.y;
                            pixelYX[p].y += phasor.y * visYX.x;

                            pixelYY[p].x += phasor.x * visYY.x;
                            pixelYY[p].y += phasor.x * visYY.y;
                            pixelYY[p].x -= phasor.y * visYY.y;
                            pixelYY[p].y += phasor.y * visYY.x;
                        }
                    }
                }
            }
            for (int p = 0; p < UNROLL_PIXELS; p++)
            {

                int x = (i + p * nr_threads) % subgrid_size;
                int y = (i + p * nr_threads) / subgrid_size;

                // Load aterm for station1
                float2 aXX1, aXY1, aYX1, aYY1;
                read_aterm(subgrid_size, nr_stations, aterm_index, station1, y, x, aterms, &aXX1, &aXY1, &aYX1, &aYY1);

                // Load aterm for station2
                float2 aXX2, aXY2, aYX2, aYY2;
                read_aterm(subgrid_size, nr_stations, aterm_index, station2, y, x, aterms, &aXX2, &aXY2, &aYX2, &aYY2);

                // Apply the conjugate transpose of the A-term
                apply_aterm(conj(aXX1), conj(aYX1), conj(aXY1), conj(aYY1), conj(aXX2), conj(aYX2), conj(aXY2),
                            conj(aYY2), pixelXX[p], pixelXY[p], pixelYX[p], pixelYY[p]);

                // Load a term for station1
                // Load spheroidal
                float sph = spheroidal[y * subgrid_size + x];

                int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i + p * nr_threads);
                int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i + p * nr_threads);
                int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i + p * nr_threads);
                int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i + p * nr_threads);

                subgrids[idx_xx] += pixelXX[p] * sph;
                subgrids[idx_xy] += pixelXY[p] * sph;
                subgrids[idx_yx] += pixelYX[p] * sph;
                subgrids[idx_yy] += pixelYY[p] * sph;
            }
            // }
        }
    }

#define KERNEL_GRIDDER_TEMPLATE(current_nr_channels)                                                                   \
    for (; (channel_offset + current_nr_channels) <= nr_channels; channel_offset += current_nr_channels)               \
    {                                                                                                                  \
        kernel_gridder_v6_<current_nr_channels>(grid_size, subgrid_size, image_size, w_step_in_lambda, nr_channels,    \
                                                channel_offset, nr_stations, uvw, wavenumbers, visibilities,           \
                                                spheroidal, aterms, metadata, subgrids);                               \
    }

    __global__ void

    kernel_gridder(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                   int nr_channels, // channel_offset? for the macro?
                   int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers, float2 *visibilities,
                   float *spheroidal, float2 *aterms, Metadata *metadata, float2 *subgrids)
    {
        int channel_offset = 0;
        KERNEL_GRIDDER_TEMPLATE(8);
        KERNEL_GRIDDER_TEMPLATE(7);
        KERNEL_GRIDDER_TEMPLATE(6);
        KERNEL_GRIDDER_TEMPLATE(5);
        KERNEL_GRIDDER_TEMPLATE(4);
        KERNEL_GRIDDER_TEMPLATE(3);
        KERNEL_GRIDDER_TEMPLATE(2);
        KERNEL_GRIDDER_TEMPLATE(1);
    }

#endif

#ifdef V7
    __shared__ float4 visibilities_v7_[BATCH_SIZE][2];
    __shared__ float4 uvw_v7_[BATCH_SIZE];
    __shared__ float wavenumbers_v7_[MAX_NR_CHANNELS];

    template <int current_nr_channels>
    __device__ void kernel_gridder_v7_(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                                       int nchannels,
                                       int channel_offset, // channel_offset? for the macro?
                                       int nstations, const UVWCoordinate<float> *uvw, const float *wavenumbers,
                                       const float2 *visibilities, const float *spheroidal, const float2 *aterms,
                                       const Metadata *metadata, float2 *subgrids)
    {
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        int tid = tidx + tidy * blockDim.x;
        int nr_threads = blockDim.x * blockDim.y;
        int s = blockIdx.x;

        // Find offset of first subgrid
        const Metadata m_0 = metadata[0];
        const int baseline_offset_1 = m_0.baseline_offset;

        // Load metadata
        const Metadata m = metadata[s];
        const int time_offset_global = (m.baseline_offset - baseline_offset_1) + m.time_offset;
        const int nr_timesteps = m.nr_timesteps;
        const int aterm_index = m.aterm_index;
        const int station1 = m.baseline.station1;
        const int station2 = m.baseline.station2;
        const int x_coordinate = m.coordinate.x;
        const int y_coordinate = m.coordinate.y;
        const float w_offset_in_lambda = w_step_in_lambda * (m.coordinate.z + 0.5);

        // Set subgrid to zero
        if (channel_offset == 0)
        {
            for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads)
            {
                int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i);
                int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i);
                int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i);
                int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i);
                subgrids[idx_xx] = make_float2(0, 0);
                subgrids[idx_xy] = make_float2(0, 0);
                subgrids[idx_yx] = make_float2(0, 0);
                subgrids[idx_yy] = make_float2(0, 0);
            }
        }

        for (int i = tid; i < current_nr_channels; i += nr_threads)
        {
            wavenumbers_v7_[i] = wavenumbers[i + channel_offset];
        }

        __syncthreads();

        // Compute u and v offset in wavelenghts
        const float u_offset = (x_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float v_offset = (y_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float w_offset = 2 * M_PI * w_offset_in_lambda;

        // Iterate all pixels in subgrid
        for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads * UNROLL_PIXELS)
        {
            float2 pixelXX[UNROLL_PIXELS];
            float2 pixelXY[UNROLL_PIXELS];
            float2 pixelYX[UNROLL_PIXELS];
            float2 pixelYY[UNROLL_PIXELS];

            for (int p = 0; p < UNROLL_PIXELS; p++)
            {
                pixelXX[p] = make_float2(0, 0);
                pixelXY[p] = make_float2(0, 0);
                pixelYX[p] = make_float2(0, 0);
                pixelYY[p] = make_float2(0, 0);
            }

            float l[UNROLL_PIXELS];
            float m[UNROLL_PIXELS];
            float n[UNROLL_PIXELS];
            float phase_offset[UNROLL_PIXELS];
            for (int p = 0; p < UNROLL_PIXELS; p++)
            {

                int x = (i + p * nr_threads) % subgrid_size;
                int y = (i + p * nr_threads) / subgrid_size;
                l[p] = compute_l(x, subgrid_size, image_size);
                m[p] = compute_m(y, subgrid_size, image_size);
                n[p] = compute_n(l[p], m[p]);
                phase_offset[p] = u_offset * l[p] + v_offset * m[p] + w_offset * n[p];
            }

            int current_nr_timesteps = BATCH_SIZE / MAX_NR_CHANNELS;
            // Iterate all timesteps
            for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps)
            {
                current_nr_timesteps = nr_timesteps - time_offset_local < current_nr_timesteps
                                           ? nr_timesteps - time_offset_local
                                           : current_nr_timesteps;
                __syncthreads();
                for (int time = tid; time < current_nr_timesteps; time += nr_threads)
                {
                    UVWCoordinate<float> a = uvw[time_offset_global + time_offset_local + time];
                    uvw_v7_[time] = make_float4(a.u, a.v, a.w, 0);
                }

                // Load visibilities
                for (int ii = tid; ii < current_nr_timesteps * current_nr_channels * 2; ii += nr_threads)
                {
                    int j = ii % 2; // one thread loads either upper or lower float4 part of visibility
                    int k = ii / 2;
                    int idx_time = time_offset_global + time_offset_local + (k / current_nr_channels);
                    int idx_chan = channel_offset + (k % current_nr_channels);
                    int idx_vis = index_visibility(nchannels, idx_time, idx_chan, 0);
                    float4 *vis_ptr = (float4 *)&visibilities[idx_vis];
                    visibilities_v7_[k][j] = vis_ptr[j];
                }

                __syncthreads();

                for (int time = 0; time < current_nr_timesteps; time++)
                {

                    // Load UVW coordinates
                    float u = uvw_v7_[time].x;
                    float v = uvw_v7_[time].y;
                    float w = uvw_v7_[time].z;
                    for (int p = 0; p < UNROLL_PIXELS; p++)
                    {
                        // Compute phase index
                        float phase_index = u * l[p] + v * m[p] + w * n[p];

                        for (int chan = 0; chan < current_nr_channels; chan++)
                        {
                            // Compute phase
                            float phase = phase_offset[p] - (phase_index * wavenumbers_v7_[chan]);

                            // Compute phasor
                            float2 phasor = make_float2(__cosf(phase), __sinf(phase));

                            // Update pixel for every polarization

                            // Load visibilities from shared memory
                            float4 a = visibilities_v7_[time * current_nr_channels + chan][0];
                            float4 b = visibilities_v7_[time * current_nr_channels + chan][1];
                            float2 visXX = make_float2(a.x, a.y);
                            float2 visXY = make_float2(a.z, a.w);
                            float2 visYX = make_float2(b.x, b.y);
                            float2 visYY = make_float2(b.z, b.w);

                            pixelXX[p].x += phasor.x * visXX.x;
                            pixelXX[p].y += phasor.x * visXX.y;
                            pixelXX[p].x -= phasor.y * visXX.y;
                            pixelXX[p].y += phasor.y * visXX.x;

                            pixelXY[p].x += phasor.x * visXY.x;
                            pixelXY[p].y += phasor.x * visXY.y;
                            pixelXY[p].x -= phasor.y * visXY.y;
                            pixelXY[p].y += phasor.y * visXY.x;

                            pixelYX[p].x += phasor.x * visYX.x;
                            pixelYX[p].y += phasor.x * visYX.y;
                            pixelYX[p].x -= phasor.y * visYX.y;
                            pixelYX[p].y += phasor.y * visYX.x;

                            pixelYY[p].x += phasor.x * visYY.x;
                            pixelYY[p].y += phasor.x * visYY.y;
                            pixelYY[p].x -= phasor.y * visYY.y;
                            pixelYY[p].y += phasor.y * visYY.x;
                        }
                    }
                }
            }

            for (int p = 0; p < UNROLL_PIXELS; p++)
            {
                int x = (i + p * nr_threads) % subgrid_size;
                int y = (i + p * nr_threads) / subgrid_size;

                // Load aterm for station1
                float2 aXX1, aXY1, aYX1, aYY1;
                read_aterm(subgrid_size, nstations, aterm_index, station1, y, x, aterms, &aXX1, &aXY1, &aYX1, &aYY1);

                // Load aterm for station2
                float2 aXX2, aXY2, aYX2, aYY2;
                read_aterm(subgrid_size, nstations, aterm_index, station2, y, x, aterms, &aXX2, &aXY2, &aYX2, &aYY2);

                // Apply the conjugate transpose of the A-term
                apply_aterm(conj(aXX1), conj(aYX1), conj(aXY1), conj(aYY1), conj(aXX2), conj(aYX2), conj(aXY2),
                            conj(aYY2), pixelXX[p], pixelXY[p], pixelYX[p], pixelYY[p]);

                // Load a term for station1
                // Load spheroidal
                float sph = spheroidal[y * subgrid_size + x];

                int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i + p * nr_threads);
                int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i + p * nr_threads);
                int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i + p * nr_threads);
                int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i + p * nr_threads);

                subgrids[idx_xx] += pixelXX[p] * sph;
                subgrids[idx_xy] += pixelXY[p] * sph;
                subgrids[idx_yx] += pixelYX[p] * sph;
                subgrids[idx_yy] += pixelYY[p] * sph;
            }
        }
    }

#define KERNEL_GRIDDER_TEMPLATE(current_nr_channels)                                                                   \
    for (; (channel_offset + current_nr_channels) <= nr_channels; channel_offset += current_nr_channels)               \
    {                                                                                                                  \
        kernel_gridder_v7_<current_nr_channels>(grid_size, subgrid_size, image_size, w_step_in_lambda, nr_channels,    \
                                                channel_offset, nr_stations, uvw, wavenumbers, visibilities,           \
                                                spheroidal, aterms, metadata, subgrids);                               \
    }

    __global__ void kernel_gridder(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                                   int nr_channels, // channel_offset? for the macro?
                                   int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers, float2 *visibilities,
                                   float *spheroidal, float2 *aterms, Metadata *metadata, float2 *subgrids)
    {
        int channel_offset = 0;
        KERNEL_GRIDDER_TEMPLATE(8);
        KERNEL_GRIDDER_TEMPLATE(7);
        KERNEL_GRIDDER_TEMPLATE(6);
        KERNEL_GRIDDER_TEMPLATE(5);
        KERNEL_GRIDDER_TEMPLATE(4);
        KERNEL_GRIDDER_TEMPLATE(3);
        KERNEL_GRIDDER_TEMPLATE(2);
        KERNEL_GRIDDER_TEMPLATE(1);
    }
#endif

#ifdef V8
    __shared__ float4 visibilities_v8_[BATCH_SIZE][2];
    __shared__ float4 uvw_v8_[BATCH_SIZE];
    __shared__ float wavenumbers_v8_[MAX_NR_CHANNELS];

    template <int current_nr_channels>
    __device__ void kernel_gridder_v8_(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                                       int nr_channels,
                                       int channel_offset, // channel_offset? for the macro?
                                       int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers,
                                       float2 *visibilities, float *spheroidal, float2 *aterms, Metadata *metadata,
                                       float2 *subgrids)
    {
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        int tid = tidx + tidy * blockDim.x;
        int nr_threads = blockDim.x * blockDim.y;
        int s = blockIdx.x;

        // Find offset of first subgrid
        const Metadata m_0 = metadata[0];
        const int baseline_offset_1 = m_0.baseline_offset;

        // Load metadata
        const Metadata m = metadata[s];
        const int time_offset_global = (m.baseline_offset - baseline_offset_1) + m.time_offset;
        const int nr_timesteps = m.nr_timesteps;
        const int aterm_index = m.aterm_index;
        const int station1 = m.baseline.station1;
        const int station2 = m.baseline.station2;
        const int x_coordinate = m.coordinate.x;
        const int y_coordinate = m.coordinate.y;
        const float w_offset_in_lambda = w_step_in_lambda * (m.coordinate.z + 0.5);

        // Set subgrid to zero
        if (channel_offset == 0)
        {
            for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads)
            {
                int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i);
                int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i);
                int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i);
                int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i);
                subgrids[idx_xx] = make_float2(0, 0);
                subgrids[idx_xy] = make_float2(0, 0);
                subgrids[idx_yx] = make_float2(0, 0);
                subgrids[idx_yy] = make_float2(0, 0);
            }
        }

        for (int i = tid; i < current_nr_channels; i += nr_threads)
        {
            wavenumbers_v8_[i] = wavenumbers[i + channel_offset];
        }

        __syncthreads();

        // Compute u and v offset in wavelenghts
        const float u_offset = (x_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float v_offset = (y_coordinate + subgrid_size / 2 - grid_size / 2) * (2 * M_PI / image_size);
        const float w_offset = 2 * M_PI * w_offset_in_lambda;

        // Iterate all pixels in subgrid
        for (int i = tid; i < subgrid_size * subgrid_size; i += nr_threads * UNROLL_PIXELS)
        {
            // for (int x = 0; x < subgrid_size; x++) {
            //  Initialize pixel for every polarization
            // int x = i % subgrid_size;
            // int y = i / subgrid_size;

            float2 pixelXX[UNROLL_PIXELS];
            float2 pixelXY[UNROLL_PIXELS];
            float2 pixelYX[UNROLL_PIXELS];
            float2 pixelYY[UNROLL_PIXELS];

            for (int p = 0; p < UNROLL_PIXELS; p++)
            {
                pixelXX[p] = make_float2(0, 0);
                pixelXY[p] = make_float2(0, 0);
                pixelYX[p] = make_float2(0, 0);
                pixelYY[p] = make_float2(0, 0);
            }

            float l[UNROLL_PIXELS];
            float m[UNROLL_PIXELS];
            float n[UNROLL_PIXELS];
            float phase_offset[UNROLL_PIXELS];
            for (int p = 0; p < UNROLL_PIXELS; p++)
            {

                int x = (i + p * nr_threads) % subgrid_size;
                int y = (i + p * nr_threads) / subgrid_size;
                l[p] = compute_l(x, subgrid_size, image_size);
                m[p] = compute_m(y, subgrid_size, image_size);
                n[p] = compute_n(l[p], m[p]);
                phase_offset[p] = u_offset * l[p] + v_offset * m[p] + w_offset * n[p];
            }

            int current_nr_timesteps = BATCH_SIZE / MAX_NR_CHANNELS;
            // Iterate all timesteps
            for (int time_offset_local = 0; time_offset_local < nr_timesteps; time_offset_local += current_nr_timesteps)
            {
                current_nr_timesteps = nr_timesteps - time_offset_local < current_nr_timesteps
                                           ? nr_timesteps - time_offset_local
                                           : current_nr_timesteps;
                __syncthreads();
                for (int time = tid; time < current_nr_timesteps; time += nr_threads)
                {
                    UVWCoordinate<float> a = uvw[time_offset_global + time_offset_local + time];
                    uvw_v8_[time] = make_float4(a.u, a.v, a.w, 0);
                }

                // Load visibilities
                for (int ii = tid; ii < current_nr_timesteps * current_nr_channels * 2; ii += nr_threads)
                {
                    int j = ii % 2; // one thread loads either upper or lower float4 part of visibility
                    int k = ii / 2;
                    int idx_time = time_offset_global + time_offset_local + (k / current_nr_channels);
                    int idx_chan = channel_offset + (k % current_nr_channels);
                    int idx_vis = index_visibility(nr_channels, idx_time, idx_chan, 0);
                    float4 *vis_ptr = (float4 *)&visibilities[idx_vis];
                    visibilities_v8_[k][j] = vis_ptr[j];
                }

                __syncthreads();

                for (int time = 0; time < current_nr_timesteps; time++)
                {

                    // Load UVW coordinates
                    float u = uvw_v8_[time].x;
                    float v = uvw_v8_[time].y;
                    float w = uvw_v8_[time].z;

                    float phase_index[UNROLL_PIXELS];
                    for (int p = 0; p < UNROLL_PIXELS; p++)
                    {
                        phase_index[p] = u * l[p] + v * m[p] + w * n[p];
                    }

                    float2 phasor_d[UNROLL_PIXELS];
                    float2 phasor_c[UNROLL_PIXELS];
                    for (int p = 0; p < UNROLL_PIXELS; p++)
                    {
                        float phase_0 = phase_offset[p] - (phase_index[p] * wavenumbers_v8_[0]);
                        float phase_1 = phase_offset[p] - (phase_index[p] * wavenumbers_v8_[current_nr_channels - 1]);
                        float phase_d = phase_1 - phase_0;
                        if (current_nr_channels > 1)
                        {
                            phase_d *= 1.0f / (current_nr_channels - 1);
                        }
                        __sincosf(phase_0, &phasor_c[p].y, &phasor_c[p].x);
                        __sincosf(phase_d, &phasor_d[p].y, &phasor_d[p].x);
                    }

                    for (int chan = 0; chan < current_nr_channels; chan++)
                    {
                        float4 a = visibilities_v8_[time * current_nr_channels + chan][0];
                        float4 b = visibilities_v8_[time * current_nr_channels + chan][1];
                        float2 visXX = make_float2(a.x, a.y);
                        float2 visXY = make_float2(a.z, a.w);
                        float2 visYX = make_float2(b.x, b.y);
                        float2 visYY = make_float2(b.z, b.w);
                        for (int p = 0; p < UNROLL_PIXELS; p++)
                        {
                            float2 phasor = phasor_c[p];

                            pixelXX[p].x += phasor.x * visXX.x;
                            pixelXX[p].y += phasor.x * visXX.y;
                            pixelXX[p].x -= phasor.y * visXX.y;
                            pixelXX[p].y += phasor.y * visXX.x;

                            pixelXY[p].x += phasor.x * visXY.x;
                            pixelXY[p].y += phasor.x * visXY.y;
                            pixelXY[p].x -= phasor.y * visXY.y;
                            pixelXY[p].y += phasor.y * visXY.x;

                            pixelYX[p].x += phasor.x * visYX.x;
                            pixelYX[p].y += phasor.x * visYX.y;
                            pixelYX[p].x -= phasor.y * visYX.y;
                            pixelYX[p].y += phasor.y * visYX.x;

                            pixelYY[p].x += phasor.x * visYY.x;
                            pixelYY[p].y += phasor.x * visYY.y;
                            pixelYY[p].x -= phasor.y * visYY.y;
                            pixelYY[p].y += phasor.y * visYY.x;

                            if (chan < current_nr_channels - 1)
                            {
                                phasor_c[p] = phasor_c[p] * phasor_d[p];
                            }
                        }
                    }

                    /*
                    for (int p = 0; p < UNROLL_PIXELS; p++) {

                      for (int chan = 0; chan < current_nr_channels; chan++) {
                        // Compute phase
                        float phase =
                            phase_offset[p] - (phase_index[p] * wavenumbers_v8_[chan]);

                        // Compute phasor
                        float2 phasor = make_float2(__cosf(phase), __sinf(phase));

                        // Update pixel for every polarization

                      // Load visibilities from shared memory
                                float4 a = visibilities_v8_[time*current_nr_channels+chan][0];
                                float4 b = visibilities_v8_[time*current_nr_channels+chan][1];
                                float2 visXX = make_float2(a.x, a.y);
                                float2 visXY = make_float2(a.z, a.w);
                                float2 visYX = make_float2(b.x, b.y);
                                float2 visYY = make_float2(b.z, b.w);


                        pixelXX[p].x += phasor.x * visXX.x;
                        pixelXX[p].y += phasor.x * visXX.y;
                        pixelXX[p].x -= phasor.y * visXX.y;
                        pixelXX[p].y += phasor.y * visXX.x;

                        pixelXY[p].x += phasor.x * visXY.x;
                        pixelXY[p].y += phasor.x * visXY.y;
                        pixelXY[p].x -= phasor.y * visXY.y;
                        pixelXY[p].y += phasor.y * visXY.x;

                        pixelYX[p].x += phasor.x * visYX.x;
                        pixelYX[p].y += phasor.x * visYX.y;
                        pixelYX[p].x -= phasor.y * visYX.y;
                        pixelYX[p].y += phasor.y * visYX.x;

                        pixelYY[p].x += phasor.x * visYY.x;
                        pixelYY[p].y += phasor.x * visYY.y;
                        pixelYY[p].x -= phasor.y * visYY.y;
                        pixelYY[p].y += phasor.y * visYY.x;
                      }
                    }*/
                }
            }
            for (int p = 0; p < UNROLL_PIXELS; p++)
            {

                int x = (i + p * nr_threads) % subgrid_size;
                int y = (i + p * nr_threads) / subgrid_size;

                // Load aterm for station1
                float2 aXX1, aXY1, aYX1, aYY1;
                read_aterm(subgrid_size, nr_stations, aterm_index, station1, y, x, aterms, &aXX1, &aXY1, &aYX1, &aYY1);

                // Load aterm for station2
                float2 aXX2, aXY2, aYX2, aYY2;
                read_aterm(subgrid_size, nr_stations, aterm_index, station2, y, x, aterms, &aXX2, &aXY2, &aYX2, &aYY2);

                // Apply the conjugate transpose of the A-term
                apply_aterm(conj(aXX1), conj(aYX1), conj(aXY1), conj(aYY1), conj(aXX2), conj(aYX2), conj(aXY2),
                            conj(aYY2), pixelXX[p], pixelXY[p], pixelYX[p], pixelYY[p]);

                // Load a term for station1
                // Load spheroidal
                float sph = spheroidal[y * subgrid_size + x];

                int idx_xx = index_subgrid_(subgrid_size, s, 0, 0, i + p * nr_threads);
                int idx_xy = index_subgrid_(subgrid_size, s, 1, 0, i + p * nr_threads);
                int idx_yx = index_subgrid_(subgrid_size, s, 2, 0, i + p * nr_threads);
                int idx_yy = index_subgrid_(subgrid_size, s, 3, 0, i + p * nr_threads);

                subgrids[idx_xx] += pixelXX[p] * sph;
                subgrids[idx_xy] += pixelXY[p] * sph;
                subgrids[idx_yx] += pixelYX[p] * sph;
                subgrids[idx_yy] += pixelYY[p] * sph;
            }
            // }
        }
    }

#define KERNEL_GRIDDER_TEMPLATE(current_nr_channels)                                                                   \
    for (; (channel_offset + current_nr_channels) <= nr_channels; channel_offset += current_nr_channels)               \
    {                                                                                                                  \
        kernel_gridder_v8_<current_nr_channels>(grid_size, subgrid_size, image_size, w_step_in_lambda, nr_channels,    \
                                                channel_offset, nr_stations, uvw, wavenumbers, visibilities,           \
                                                spheroidal, aterms, metadata, subgrids);                               \
    }

    __global__ void

    kernel_gridder(const int grid_size, int subgrid_size, float image_size, float w_step_in_lambda,
                   int nr_channels, // channel_offset? for the macro?
                   int nr_stations, UVWCoordinate<float> *uvw, float *wavenumbers, float2 *visibilities,
                   float *spheroidal, float2 *aterms, Metadata *metadata, float2 *subgrids)
    {
        int channel_offset = 0;
        KERNEL_GRIDDER_TEMPLATE(8);
        KERNEL_GRIDDER_TEMPLATE(7);
        KERNEL_GRIDDER_TEMPLATE(6);
        KERNEL_GRIDDER_TEMPLATE(5);
        KERNEL_GRIDDER_TEMPLATE(4);
        KERNEL_GRIDDER_TEMPLATE(3);
        KERNEL_GRIDDER_TEMPLATE(2);
        KERNEL_GRIDDER_TEMPLATE(1);
    }

#endif

    template <>
    benchmark_result gridding_benchmark_launcher<benchmarks_common::hardware_type::gpu>::launch(
        const gridding_benchmark::configuration &configuration, Array2D<UVWCoordinate<float>> &uvw,
        Array3D<Visibility<std::complex<float>>> &visibilities, Array1D<Baseline> &baselines,
        Array4D<Matrix2x2<std::complex<float>>> &aterms, Array1D<float> &frequencies, Array1D<float> &wavenumbers,
        Array2D<float> &spheroidal, Array4D<std::complex<float>> &subgrids, Array1D<Metadata> &metadata)
    {
        size_t total_in_size = 0;

        UVWCoordinate<float> *d_uvw;
        const auto d_uvw_size = uvw.size() * sizeof(uvw[0]);
        total_in_size += d_uvw_size;
        cudaCheck(hipMalloc(&d_uvw, d_uvw_size));

        float *d_wavenumbers;
        const auto d_wavenumbers_size = wavenumbers.size() * sizeof(wavenumbers[0]);
        total_in_size += d_wavenumbers_size;
        cudaCheck(hipMalloc(&d_wavenumbers, d_wavenumbers_size));

        float2 *d_visibilities;
        const auto d_visibilities_size = visibilities.size() * sizeof(visibilities[0]);
        total_in_size += d_visibilities_size;
        cudaCheck(hipMalloc(&d_visibilities, d_visibilities_size));

        float *d_spheroidal;
        const auto d_spheroidal_size = spheroidal.size() * sizeof(spheroidal[0]);
        total_in_size += d_spheroidal_size;
        cudaCheck(hipMalloc(&d_spheroidal, d_spheroidal_size));

        float2 *d_aterms;
        const auto d_aterms_size = aterms.size() * sizeof(aterms[0]);
        total_in_size += d_aterms_size;
        cudaCheck(hipMalloc(&d_aterms, d_aterms_size));

        Metadata *d_metadata;
        const auto d_metadata_size = metadata.size() * sizeof(metadata[0]);
        total_in_size += d_metadata_size;
        cudaCheck(hipMalloc(&d_metadata, d_metadata_size));

        float2 *d_subgrids;
        const auto d_subgrids_size = subgrids.size() * sizeof(subgrids[0]);
        cudaCheck(hipMalloc(&d_subgrids, d_subgrids_size));
        const auto total_out_size = d_subgrids_size;

        hipEvent_t begin_in_memcpy;
        cudaCheck(hipEventCreate(&begin_in_memcpy));
        hipEvent_t end_in_memcpy;
        cudaCheck(hipEventCreate(&end_in_memcpy));

        cudaCheck(hipEventRecord(begin_in_memcpy));
        for (size_t i = 0; i < configuration.niterations; ++i)
        {
            cudaCheck(hipMemcpy(d_uvw, uvw.data(), d_uvw_size, hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(d_wavenumbers, wavenumbers.data(), d_wavenumbers_size, hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(d_visibilities, visibilities.data(), d_visibilities_size, hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(d_spheroidal, spheroidal.data(), d_spheroidal_size, hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(d_aterms, aterms.data(), d_aterms_size, hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(d_metadata, metadata.data(), d_metadata_size, hipMemcpyHostToDevice));
            cudaCheck(hipEventRecord(end_in_memcpy));
        }
        cudaCheck(hipEventSynchronize(end_in_memcpy));

        float in_copy_time_ms;
        cudaCheck(hipEventElapsedTime(&in_copy_time_ms, begin_in_memcpy, end_in_memcpy));
        in_copy_time_ms /= static_cast<float>(configuration.niterations);

        const auto n_baselines = (configuration.nstations * (configuration.nstations - 1)) / 2;
        const auto n_subgrids = n_baselines * configuration.ntimeslots;
        const auto w_step_in_lambda = 0;
        const auto image_size = 0.1F;

        hipEvent_t begin_compute;
        cudaCheck(hipEventCreate(&begin_compute));
        hipEvent_t end_compute;
        cudaCheck(hipEventCreate(&end_compute));

            kernel_gridder<<<(n_subgrids + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(
                configuration.grid_size, configuration.subgrid_size, image_size, w_step_in_lambda,
                configuration.nchannels, configuration.nstations, d_uvw, d_wavenumbers, d_visibilities, d_spheroidal,
                d_aterms, d_metadata, d_subgrids);
                hipDeviceSynchronize();

        cudaCheck(hipEventRecord(begin_compute));
        for (size_t i = 0; i < configuration.niterations; ++i)
        {
            kernel_gridder<<<(n_subgrids + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(
                configuration.grid_size, configuration.subgrid_size, image_size, w_step_in_lambda,
                configuration.nchannels, configuration.nstations, d_uvw, d_wavenumbers, d_visibilities, d_spheroidal,
                d_aterms, d_metadata, d_subgrids);
        }
        cudaCheck(hipEventRecord(end_compute));
        cudaCheck(hipEventSynchronize(end_compute));

        float compute_time_ms;
        cudaCheck(hipEventElapsedTime(&compute_time_ms, begin_compute, end_compute));

        hipEvent_t begin_out_memcpy;
        cudaCheck(hipEventCreate(&begin_out_memcpy));
        hipEvent_t end_out_memcpy;
        cudaCheck(hipEventCreate(&end_out_memcpy));

        cudaCheck(hipEventRecord(begin_out_memcpy));
        for (size_t i = 0; i < configuration.niterations; ++i)
        {
            cudaCheck(hipMemcpy(subgrids.data(), d_subgrids, d_subgrids_size, hipMemcpyDeviceToHost));
        }
        cudaCheck(hipEventRecord(end_out_memcpy));
        cudaCheck(hipEventSynchronize(end_out_memcpy));

        float out_copy_time_ms;
        cudaCheck(hipEventElapsedTime(&out_copy_time_ms, begin_out_memcpy, end_out_memcpy));
        out_copy_time_ms /= static_cast<float>(configuration.niterations);

        benchmark_result result;
        result.in_transfer_time = in_copy_time_ms * 1000;
        result.out_transfer_time = out_copy_time_ms * 1000;
        result.in_bandwidth = static_cast<float>(total_in_size) / (1000.f * in_copy_time_ms);
        result.out_bandwidth = static_cast<float>(total_out_size) / (1000.f * out_copy_time_ms);
        result.compute_time = compute_time_ms * 1000 / static_cast<float>(configuration.niterations);

        return result;
    }
} // namespace gridding_benchmark
